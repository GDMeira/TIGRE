#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * CUDA functions for ray-voxel intersection based projection
 *
 * This file has the necesary fucntiosn to perform X-ray CBCT projection
 * operation given a geaometry, angles and image. It usesthe so-called
 * Jacobs algorithm to compute efficiently the length of the x-rays over
 * voxel space.
 *
 * CODE by       Ander Biguri
 *               Sepideh Hatamikia (arbitrary rotation)
 * ---------------------------------------------------------------------------
 * ---------------------------------------------------------------------------
 * Copyright (c) 2015, University of Bath and CERN- European Organization for
 * Nuclear Research
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its contributors
 * may be used to endorse or promote products derived from this software without
 * specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * ---------------------------------------------------------------------------
 *
 * Contact: tigre.toolbox@gmail.com
 * Codes  : https://github.com/CERN/TIGRE
 * ---------------------------------------------------------------------------
 */

#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "Siddon_projection.hpp"
#include "TIGRE_common.hpp"
#include <math.h>

#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
                mexPrintf("%s \n",msg);\
                mexErrMsgIdAndTxt("Ax:Siddon_projection",hipGetErrorString(__err));\
        } \
} while (0)
    
    
#define MAXTREADS 1024
#define PROJ_PER_BLOCK 9
#define PIXEL_SIZE_BLOCK 9
    /*GEOMETRY DEFINITION
     *
     *                Detector plane, behind
     *            |-----------------------------|
     *            |                             |
     *            |                             |
     *            |                             |
     *            |                             |
     *            |      +--------+             |
     *            |     /        /|             |
     *   A Z      |    /        / |*D           |
     *   |        |   +--------+  |             |
     *   |        |   |        |  |             |
     *   |        |   |     *O |  +             |
     *    --->y   |   |        | /              |
     *  /         |   |        |/               |
     * V X        |   +--------+                |
     *            |-----------------------------|
     *
     *           *S
     *
     *
     *
     *
     *
     **/
    
    void CreateTexture(const GpuIds& gpuids,const float* imagedata,Geometry geo,hipArray** d_cuArrTex, hipTextureObject_t *texImage,bool alloc);

__constant__ Point3D projParamsArrayDev[6*PROJ_PER_BLOCK];  // Dev means it is on device


__global__ void vecAddInPlace(float *a, float *b, unsigned long  n)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    // Make sure we do not go out of bounds
    if (idx < n)
        a[idx] = a[idx] + b[idx];
}

__global__ void kernelPixelDetector( Geometry geo,
        float* detector,
        const int currProjSetNumber,
        const int totalNoOfProjections,
        hipTextureObject_t tex){
    
    unsigned long long u = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long v = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned long long projNumber=threadIdx.z;
    unsigned long long rayNumber=blockIdx.z;

    float gelTubeRadius = projParamsArrayDev[6 * projNumber + 5].x; //aux2.x;
    u += (unsigned long long)geo.nVoxelY/2 - gelTubeRadius / geo.dVoxelY;
    
    float minU = geo.nVoxelY/2 - gelTubeRadius / geo.dVoxelY;
    float maxU = geo.nVoxelY/2 + gelTubeRadius / geo.dVoxelY;
    
    if (u>maxU ||v>= geo.nDetecV || projNumber>=PROJ_PER_BLOCK)
        return;
    
#if IS_FOR_MATLAB_TIGRE
    size_t idx =  (size_t)(u * (unsigned long long)geo.nDetecV + v)+ projNumber*(unsigned long long)geo.nDetecV *(unsigned long long)geo.nDetecU ;
#else
    size_t idx =  (size_t)(v * (unsigned long long)geo.nDetecU + u)+ projNumber*(unsigned long long)geo.nDetecV *(unsigned long long)geo.nDetecU ;
#endif
    unsigned long indAlpha = currProjSetNumber*PROJ_PER_BLOCK+projNumber;  // This is the ABSOLUTE projection number in the projection array (for a given GPU)

    if(indAlpha>=totalNoOfProjections)
        return;
    
    Point3D uvOrigin = projParamsArrayDev[6 * projNumber];  // 6*projNumber because we have 6 Point3D values per projection
    Point3D deltaU = projParamsArrayDev[6 * projNumber + 1];
    Point3D deltaV = projParamsArrayDev[6 * projNumber + 2];
    Point3D source = projParamsArrayDev[6 * projNumber + 3];
    Point3D auxParams1 = projParamsArrayDev[6 * projNumber + 4];
    Point3D auxParams2 = projParamsArrayDev[6 * projNumber + 5];

    // float gelTubeRadius = auxParams2.x/geo.dVoxelX;
    gelTubeRadius = gelTubeRadius / geo.dVoxelX;
    float gelTubeRadiusSquare = gelTubeRadius * gelTubeRadius;
    float DSD = auxParams1.x;
    float DSO = auxParams1.y;
    float EPS = auxParams1.z;
    float h = geo.dDetecU * (DSD - DSO) / (EPS - geo.dDetecU);
    float fGrowth = EPS * (DSD + h) / (DSD - DSO + h);

            float pixelU, pixelV;

            pixelU = u;
            pixelV = geo.nDetecV-v-1;

            // float uiHC = 2;
            // float pixelU = u + (uiHC ) / (10*2);
            // float pixelV = geo.nDetecV-v-1;

            Point3D pixel1D;
            pixel1D.x=(uvOrigin.x+pixelU*deltaU.x+pixelV*deltaV.x);
            pixel1D.y=(uvOrigin.y+pixelU*deltaU.y+pixelV*deltaV.y);
            pixel1D.z=(uvOrigin.z+pixelU*deltaU.z+pixelV*deltaV.z);

            // changing source for each u, v
            // o (0,0,0) esta no canto da imagem, uvorigem eh o canto do detector nessa geometria

            float pixelUs = pixelU - (float)(geo.nDetecU - 1) * 0.5f;
            float pixelVs = pixelV - (float)(geo.nDetecV - 1) * 0.5f; 

            //Point3D deltaUd, deltaVd;
            //deltaUd.x = deltaU.x / geo.dDetecU * fGrowth;
            //deltaUd.y = deltaU.y / geo.dDetecU * fGrowth;
            //deltaUd.z = deltaU.z / geo.dDetecU * fGrowth;

            //deltaVd.x = deltaV.x / geo.dDetecV * fGrowth;
            //deltaVd.y = deltaV.y / geo.dDetecV * fGrowth;
            //deltaVd.z = deltaV.z / geo.dDetecV * fGrowth;

            //source.x += pixelUs * deltaUd.x + pixelVs * deltaVd.x;
            //source.y += pixelUs * deltaUd.y + pixelVs * deltaVd.y;
            //source.z += pixelUs * deltaUd.z + pixelVs * deltaVd.z;
            Point3D newSource;

            newSource.x = source.x + fGrowth * ( __fdividef(pixelUs * deltaU.x, geo.dDetecU) + __fdividef(pixelVs * deltaV.x, geo.dDetecV) );
            newSource.y = source.y + fGrowth * ( __fdividef(pixelUs * deltaU.y, geo.dDetecU) + __fdividef(pixelVs * deltaV.y, geo.dDetecV) );
            newSource.z = source.z + fGrowth * ( __fdividef(pixelUs * deltaU.z, geo.dDetecU) + __fdividef(pixelVs * deltaV.z, geo.dDetecV) );

            ///////
            // Siddon's ray-voxel intersection, optimized as in doi=10.1.1.55.7516
            //////
            // Also called Jacobs algorithms
            Point3D ray;
            // vector of Xray
            ray.x=pixel1D.x - newSource.x;
            ray.y=pixel1D.y - newSource.y;
            ray.z=pixel1D.z - newSource.z;

            float eps=0.000001;
            // ray.x=(abs(ray.x)<eps)? 0 : ray.x;
            // ray.y=(abs(ray.y)<eps)? 0 : ray.y;
            // ray.z=(abs(ray.z)<eps)? 0 : ray.z;

            // C = (geo.sVoxelX/2, geo.sVoxelY/2, P.z) center of the tube with P height
            Point3D C;
            C.x = geo.nVoxelX*0.5f;
            C.y = geo.nVoxelY*0.5f;

            // float distanceToCenterInTheMiddleOfImage = (ray.y*0.5f + newSource.y - C.y) * (ray.y*0.5f + newSource.y - C.y) + (ray.x*0.5f + newSource.x - C.x) * (ray.x*0.5f + newSource.x - C.x);

            // // if ray dont pass thrugh gel tube, return 0
            // if ( distanceToCenterInTheMiddleOfImage > gelTubeRadiusSquare) {
            //     // detector[idx] = 0;
            //     return;
            // }

            Point3D Q;
            Q.x = newSource.x - C.x;
            Q.y = newSource.y - C.y;
            Q.z = 0;
            float a1, a2;

            float aux1 = 2 * Q.x * ray.x + 2 * Q.y * ray.y;
            float aux2 = ray.x * ray.x + ray.y * ray.y + ray.z * ray.z;
            float aux3 = (-gelTubeRadiusSquare + Q.x * Q.x + Q.y * Q.y);

            // no real solutions or only 1 solution
            if (aux1*aux1 - 4 * aux2 * aux3 <= 0) return;

            a1 = (-__fsqrt_rd(aux1*aux1 - 4 * aux2 * aux3) - aux1) / (2*aux2);
            a2 = (__fsqrt_rd(aux1*aux1 - 4 * aux2 * aux3) - aux1) / (2*aux2);

            // points where ray intersects the gel tube
            Point3D Q1, Q2;
            Q1.x = newSource.x + a1 * ray.x;
            Q1.y = newSource.y + a1 * ray.y;
            Q1.z = newSource.z + a1 * ray.z;
            Q2.x = newSource.x + a2 * ray.x;
            Q2.y = newSource.y + a2 * ray.y;
            Q2.z = newSource.z + a2 * ray.z;

            float increment = a1/10;
            float lastRadiusSquare = (Q1.x- C.x)*(Q1.x- C.x) + (Q1.y- C.y)*(Q1.y- C.y);
            int iterationsAfterLastChange = 0;

            for (int i = 0; i < 4000; i++) {
                Q1.x = newSource.x + a1 * ray.x;
                Q1.y = newSource.y + a1 * ray.y;
                Q1.z = newSource.z + a1 * ray.z;

                float currentRadiusSquare = (Q1.x- C.x)*(Q1.x- C.x) + (Q1.y- C.y)*(Q1.y- C.y);
                if ( abs(currentRadiusSquare - gelTubeRadiusSquare) < 1) break;

                if (abs(currentRadiusSquare - gelTubeRadiusSquare) > abs(lastRadiusSquare - gelTubeRadiusSquare)) {
                    if (iterationsAfterLastChange <= 2) { //local minimum
                        a1 = a1 - increment;
                        increment = -increment/10;
                    } else {
                        increment = -increment;
                    }

                    iterationsAfterLastChange = -1;
                }

                a1 = a1 + increment;
                lastRadiusSquare = currentRadiusSquare;
                iterationsAfterLastChange++;
            }

            // This variables are ommited because
            // bx,by,bz ={0,0,0}
            // dx,dy,dz ={1,1,1}
            // compute parameter values for x-ray parametric equation. eq(3-10)
            float axm,aym,azm;
            float axM,ayM,azM;

            // In the paper Nx= number of X planes-> Nvoxel+1
            axm=min(__fdividef(Q1.x-newSource.x,ray.x),__fdividef(Q2.x-newSource.x,ray.x));
            aym=min(__fdividef(Q1.y-newSource.y,ray.y),__fdividef(Q2.y-newSource.y,ray.y));
            azm=min(__fdividef(Q1.z-newSource.z,ray.z),__fdividef(Q2.z-newSource.z,ray.z));
            axM=max(__fdividef(Q1.x-newSource.x,ray.x),__fdividef(Q2.x-newSource.x,ray.x));
            ayM=max(__fdividef(Q1.y-newSource.y,ray.y),__fdividef(Q2.y-newSource.y,ray.y));
            azM=max(__fdividef(Q1.z-newSource.z,ray.z),__fdividef(Q2.z-newSource.z,ray.z));
            
            float am=max(max(axm,aym),azm);
            float aM=min(min(axM,ayM),azM);
            
            // line intersects voxel space ->   am<aM
            if (am>=aM){
                // detector[idx]=0;
                return;
            }
            
            // Compute max/min image INDEX for intersection eq(11-19)
            // Discussion about ternary operator in CUDA: https://stackoverflow.com/questions/7104384/in-cuda-why-is-a-b010-more-efficient-than-an-if-else-version
            float imin,imax,jmin,jmax,kmin,kmax;
            // for X
            if( newSource.x<pixel1D.x){
                imin= (newSource.x+am*ray.x);
                imax=(newSource.x+aM*ray.x);
            }else{
                imax=(newSource.x+am*ray.x);
                imin= (newSource.x+aM*ray.x);
            }
            // for Y
            if( newSource.y<pixel1D.y){
                jmin= (newSource.y+am*ray.y);
                jmax=(newSource.y+aM*ray.y);
            }else{
                jmax=(newSource.y+am*ray.y);
                jmin= (newSource.y+aM*ray.y);
            }
            // for Z
            if( newSource.z<pixel1D.z){
                kmin= (newSource.z+am*ray.z);
                kmax=(newSource.z+aM*ray.z);
            }else{
                kmax=(newSource.z+am*ray.z);
                kmin= (newSource.z+aM*ray.z);
            }
            
            // get intersection point N1. eq(20-21) [(also eq 9-10)]
            float ax,ay,az;
            ax=(newSource.x<pixel1D.x)?  __fdividef(imin-newSource.x,ray.x) :  __fdividef(imax-newSource.x,ray.x);
            ay=(newSource.y<pixel1D.y)?  __fdividef(jmin-newSource.y,ray.y) :  __fdividef(jmax-newSource.y,ray.y);
            az=(newSource.z<pixel1D.z)?  __fdividef(kmin-newSource.z,ray.z) :  __fdividef(kmax-newSource.z,ray.z);
            
            // If its Infinite (i.e. ray is perpendicular to this axis), make sure its positive
            ax=(isinf(ax))? abs(ax) : ax;
            ay=(isinf(ay))? abs(ay) : ay;
            az=(isinf(az))? abs(az) : az;
            
            // get index of first intersection. eq (26) and (19)
            float i,j,k;
            float aminc=min(min(ax,ay),az);
            i=newSource.x+ (aminc+am)*0.5f*ray.x;
            j=newSource.y+ (aminc+am)*0.5f*ray.y;
            k=newSource.z+ (aminc+am)*0.5f*ray.z;
            // Initialize
            float ac=am;
            //eq (28), unit anlges
            float axu,ayu,azu;
            axu=__frcp_rd(abs(ray.x));
            ayu=__frcp_rd(abs(ray.y));
            azu=__frcp_rd(abs(ray.z));
            // eq(29), direction of update
            float iu,ju,ku;
            iu=(newSource.x< pixel1D.x)? 1.0f : -1.0f;
            ju=(newSource.y< pixel1D.y)? 1.0f : -1.0f;
            ku=(newSource.z< pixel1D.z)? 1.0f : -1.0f;
            
            float sum=0.0f;
            float traveledLength;

            // refraction to enter tube
            float rayLength = __fsqrt_rd(ray.x*ray.x + ray.y*ray.y + ray.z*ray.z);
            Point3D v1, P, n, N, v2;

            // incident ray versor
            v1.x = __fdividef(ray.x, rayLength);
            v1.y = __fdividef(ray.y, rayLength);
            v1.z = __fdividef(ray.z, rayLength);
            
            // P = (i, j, k) refraction point
            P.x = Q1.x;
            P.y = Q1.y;
            P.z = Q1.z;

            // n = (P - C) / ||P - C|| normal vector to tube surface
            float nLength = __fsqrt_rd((P.x - C.x)*(P.x - C.x) + (P.y - C.y)*(P.y - C.y));
            n.x = (P.x - C.x) / nLength;
            n.y = (P.y - C.y) / nLength;
            n.z = 0; // (P.z - C.z) / nLength

            float incidenceAngle = acos(v1.x*n.x + v1.y*n.y + v1.z*n.z);
            incidenceAngle = (incidenceAngle > PI_2) ? PI_1 - incidenceAngle : incidenceAngle;
            float refractionAngle = asin(nWater * sin(incidenceAngle) / nGel);

            // N = n x v1 normal to the plane formed by n and v1 and v2; n.z = 0
            N.x = -n.y * v1.z;
            N.y = n.x * v1.z;
            N.z = v1.x * n.y - v1.y * n.x;

            float cos_r = cos(refractionAngle);
            float cos_ri = cos(incidenceAngle - refractionAngle);
            
            aux1 = v1.x*n.y*N.z - v1.y*n.x*N.z + v1.z * (n.x*N.y -n.y*N.x);

            v2.x = v1.x;
            v2.y = v1.y;
            v2.z = v1.z;

            // normalize v2
            float v2Length = __fsqrt_rd(v2.x*v2.x + v2.y*v2.y + v2.z*v2.z);
            v2.x = __fdividef(v2.x,  v2Length);
            v2.y = __fdividef(v2.y,  v2Length);
            v2.z = __fdividef(v2.z,  v2Length);

            float t;

            Point3D aux;
            aux.x = P.x - C.x;
            aux.y = P.y - C.y;
            aux.z = 0;

            aux1 = 2 * aux.x * v2.x + 2 * aux.y * v2.y;
            aux2 = v2.x * v2.x + v2.y * v2.y + v2.z * v2.z;
            aux3 = (-gelTubeRadiusSquare + aux.x * aux.x + aux.y * aux.y);

            // no real solutions or only 1 solution
            if (aux1*aux1 - 4 * aux2 * aux3 < 0) return;

            a1 = (-__fsqrt_rd(aux1*aux1 - 4 * aux2 * aux3) - aux1) / (2*aux2);
            a2 = (__fsqrt_rd(aux1*aux1 - 4 * aux2 * aux3) - aux1) / (2*aux2);

            t = abs(a1) > abs(a2) ? a1 : a2;
            // end of refraction
            
            Q2.x = P.x + t * v2.x;
            Q2.y = P.y + t * v2.y;
            Q2.z = P.z + t * v2.z;

            increment = t/10;
            lastRadiusSquare = (Q2.x- C.x)*(Q2.x- C.x) + (Q2.y- C.y)*(Q2.y- C.y);
            iterationsAfterLastChange = 0;

            for (int i = 0; i < 4000; i++) {
                Q2.x = P.x + t * v2.x;
                Q2.y = P.y + t * v2.y;
                Q2.z = P.z + t * v2.z;

                float currentRadiusSquare = (Q2.x- C.x)*(Q2.x- C.x) + (Q2.y- C.y)*(Q2.y- C.y);
                if ( abs(currentRadiusSquare - gelTubeRadiusSquare) < 1) break;

                if (abs(currentRadiusSquare - gelTubeRadiusSquare) > abs(lastRadiusSquare - gelTubeRadiusSquare)) {
                    if (iterationsAfterLastChange <= 2) { //local minimum
                        t = t - increment;
                        increment = -increment/10;
                    } else {
                        increment = -increment;
                    }
                    
                    iterationsAfterLastChange = -1;
                }

                t = t + increment;
                lastRadiusSquare = currentRadiusSquare;
                iterationsAfterLastChange++;
            }

            // save the ray traveled distance
            traveledLength = t;

            eps=0.000001;

            // v2*t is the distance from P to the next refraction point
            v2.x = v2.x * t;
            v2.y = v2.y * t;
            v2.z = v2.z * t;
            
            ray = v2;
            newSource = P; //current point

            Q1.x = newSource.x;
            Q1.y = newSource.y;
            Q1.z = newSource.z;
            Q2.x = newSource.x + ray.x;
            Q2.y = newSource.y + ray.y;
            Q2.z = newSource.z + ray.z;

            axm=0;
            aym=0;
            azm=0;
            axM=1;
            ayM=1;
            azM=1;
            
            // am=max(max(axm,aym),azm); // 0
            // aM=min(min(axM,ayM),azM); // 1
            am=0; // 0
            aM=1; // 1
            
            // line intersects voxel space ->   am<aM
            if (am>=aM){
                // detector[idx]=0;
                return;
            }

            if( ray.x > 0){
                imin=(am==axm)? Q1.x + 1.0f             : ceilf (newSource.x+am*ray.x);
                imax=(aM==axM)? Q2.x + 1.0f             : floorf(newSource.x+aM*ray.x);
            }else{
                imax=(am==axm)? Q1.x             : floorf(newSource.x+am*ray.x);
                imin=(aM==axM)? Q2.x             : ceilf (newSource.x+aM*ray.x);
            }
            // for Y
            if( ray.y > 0){
                jmin=(am==aym)? Q1.y + 1.0f             : ceilf (newSource.y+am*ray.y);
                jmax=(aM==ayM)? Q2.y + 1.0f             : floorf(newSource.y+aM*ray.y);
            }else{
                jmax=(am==aym)? Q1.y             : floorf(newSource.y+am*ray.y);
                jmin=(aM==ayM)? Q2.y             : ceilf (newSource.y+aM*ray.y);
            }
            // for Z
            if( ray.z > 0){
                kmin=(am==azm)? Q1.z + 1.0f             : ceilf (newSource.z+am*ray.z);
                kmax=(aM==azM)? Q2.z + 1.0f             : floorf(newSource.z+aM*ray.z);
            }else{
                kmax=(am==azm)? Q1.z             : floorf(newSource.z+am*ray.z);
                kmin=(aM==azM)? Q2.z             : ceilf (newSource.z+aM*ray.z);
            }

            // get intersection point N1. eq(20-21) [(also eq 9-10)]
            ax=(ray.x>0)?  __fdividef(imin-newSource.x,ray.x) :  __fdividef(imax-newSource.x,ray.x);
            ay=(ray.y>0)?  __fdividef(jmin-newSource.y,ray.y) :  __fdividef(jmax-newSource.y,ray.y);
            az=(ray.z>0)?  __fdividef(kmin-newSource.z,ray.z) :  __fdividef(kmax-newSource.z,ray.z);
            
            // If its Infinite (i.e. ray is perpendicular to this axis), make sure its positive
            ax=(isinf(ax))? abs(ax) : ax;
            ay=(isinf(ay))? abs(ay) : ay;
            az=(isinf(az))? abs(az) : az;    
            
            // get index of first intersection. eq (26) and (19)
            aminc=min(min(ax,ay),az);
            i=floor(newSource.x+ (aminc+am)*0.5f*ray.x);
            j=floor(newSource.y+ (aminc+am)*0.5f*ray.y);
            k=floor(newSource.z+ (aminc+am)*0.5f*ray.z);
            // Initialize
            ac=am;
            //eq (28), unit anlges
            axu=__frcp_rd(abs(ray.x));
            ayu=__frcp_rd(abs(ray.y));
            azu=__frcp_rd(abs(ray.z));

            // eq(29), direction of update
            iu=(ray.x > 0)? 1.0f : -1.0f;
            ju=(ray.y > 0)? 1.0f : -1.0f;
            ku=(ray.z > 0)? 1.0f : -1.0f;

            unsigned long Np=(abs(imax-imin)+1)+(abs(jmax-jmin)+1)+(abs(kmax-kmin)+1); // Number of intersections

            // Go iterating over the line, intersection by intersection. If double point, no worries, 0 will be computed
            i+=0.5f*iu;
            j+=0.5f*ju;
            k+=0.5f*ku;
            ax+=axu;
            ay+=ayu;
            az+=azu;
            aminc=min(min(ax,ay),az);

        #pragma unroll
            for (unsigned long interactions = 0; interactions < Np; interactions++){
                if (ax==aminc){
                    sum+=(ax-ac)*tex3D<float>(tex, i, j, k);
                    // sum+=tex3D<float>(tex, i, j, k);
                    i=i+iu;
                    ac=ax;
                    ax+=axu;
                }else if(ay==aminc){
                    sum+=(ay-ac)*tex3D<float>(tex, i, j, k);
                    // sum+=tex3D<float>(tex, i, j, k);
                    j=j+ju;
                    ac=ay;
                    ay+=ayu;
                }else if(az==aminc){
                    sum+=(az-ac)*tex3D<float>(tex, i, j, k);
                    // sum+=tex3D<float>(tex, i, j, k);
                    k=k+ku;
                    ac=az;
                    az+=azu;
                }
            
                aminc=min(min(ax,ay),az);
            }

            float resSum = sum*traveledLength*geo.dVoxelX;

            detector[idx] = resSum;
}


int siddon_ray_projection(float* img, Geometry geo, float** result,float const * const angles,int nangles, const GpuIds& gpuids){
    // Prepare for MultiGPU
    int deviceCount = gpuids.GetLength();
    cudaCheckErrors("Device query fail");
    if (deviceCount == 0) {
        mexErrMsgIdAndTxt("Ax:Siddon_projection:GPUselect","There are no available device(s) that support CUDA\n");
    }
    //
    // CODE assumes
    // 1.-All available devices are usable by this code
    // 2.-All available devices are equal, they are the same machine (warning thrown)
    // Check the available devices, and if they are the same
    if (!gpuids.AreEqualDevices()) {
        mexWarnMsgIdAndTxt("Ax:Siddon_projection:GPUselect","Detected one (or more) different GPUs.\n This code is not smart enough to separate the memory GPU wise if they have different computational times or memory limits.\n First GPU parameters used. If the code errors you might need to change the way GPU selection is performed.");
    }
    int dev;
    
    // Check free memory
    size_t mem_GPU_global;
    checkFreeMemory(gpuids, &mem_GPU_global);

    size_t mem_image=                 (unsigned long long)geo.nVoxelX*(unsigned long long)geo.nVoxelY*(unsigned long long)geo.nVoxelZ*sizeof(float);
    size_t mem_proj=                  (unsigned long long)geo.nDetecU*(unsigned long long)geo.nDetecV*sizeof(float);
    
    // Does everything fit in the GPUs?
    const bool fits_in_memory = mem_image+2*PROJ_PER_BLOCK*mem_proj<mem_GPU_global;
    const char* boolStr = fits_in_memory ? "true" : "false";
    unsigned int splits=1;
    if (!fits_in_memory) {
        // Nope nope.
        // approx free memory we have. We already have left some extra 5% free for internal stuff
        // we need a second projection memory to combine multi-GPU stuff.
        size_t mem_free=mem_GPU_global-4*PROJ_PER_BLOCK*mem_proj;
        splits=mem_image/mem_free+1;// Ceil of the truncation
    }
    Geometry* geoArray = (Geometry*)malloc(splits*sizeof(Geometry));
    splitImage(splits,geo,geoArray,nangles);
    
    // Allocate axuiliary memory for projections on the GPU to accumulate partial results
    float ** dProjection_accum;
    size_t num_bytes_proj = PROJ_PER_BLOCK*geo.nDetecU*geo.nDetecV * sizeof(float);
    if (!fits_in_memory){
        dProjection_accum=(float**)malloc(2*deviceCount*sizeof(float*));
        for (dev = 0; dev < deviceCount; dev++) {
            hipSetDevice(gpuids[dev]);
            for (int i = 0; i < 2; ++i){
                hipMalloc((void**)&dProjection_accum[dev*2+i], num_bytes_proj);
                hipMemset(dProjection_accum[dev*2+i],0,num_bytes_proj);
                cudaCheckErrors("cudaMallocauxiliarty projections fail");
            }
        }
    }
    
    // This is happening regarthless if the image fits on memory
    float** dProjection=(float**)malloc(2*deviceCount*sizeof(float*));
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(gpuids[dev]);
        
        for (int i = 0; i < 2; ++i){
            hipMalloc((void**)&dProjection[dev*2+i],   num_bytes_proj);
            hipMemset(dProjection[dev*2+i]  ,0,num_bytes_proj);
            cudaCheckErrors("hipMalloc projections fail");
        }
    }
    
    
    //Pagelock memory for synchronous copy.
    // Lets try to make the host memory pinned:
    // We laredy queried the GPU and assuemd they are the same, thus should have the same attributes.
    int isHostRegisterSupported = 0;
#if CUDART_VERSION >= 9020
    hipDeviceGetAttribute(&isHostRegisterSupported,hipDeviceAttributeHostRegisterSupported,gpuids[0]);
#endif
    // empirical testing shows that when the image split is smaller than 1 (also implies the image is not very big), the time to
    // pin the memory is greater than the lost time in Synchronously launching the memcpys. This is only worth it when the image is too big.
#ifndef NO_PINNED_MEMORY
    if (isHostRegisterSupported & (splits>1 |deviceCount>1)){
        hipHostRegister(img, (size_t)geo.nVoxelX*(size_t)geo.nVoxelY*(size_t)geo.nVoxelZ*(size_t)sizeof(float),hipHostRegisterPortable);
    }
#endif
    cudaCheckErrors("Error pinning memory");

    
    
    // auxiliary variables
    Point3D source, deltaU, deltaV, uvOrigin;
    Point3D* projParamsArrayHost;
    hipHostMalloc((void**)&projParamsArrayHost, 6 * PROJ_PER_BLOCK * sizeof(Point3D));
    cudaCheckErrors("Error allocating auxiliary constant memory");
    
    // Create Streams for overlapping memcopy and compute
    int nStreams=deviceCount*2;
    hipStream_t* stream=(hipStream_t*)malloc(nStreams*sizeof(hipStream_t));;
    
    
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(gpuids[dev]);
        for (int i = 0; i < 2; ++i){
            hipStreamCreate(&stream[i+dev*2]);
            
        }
    }
    cudaCheckErrors("Stream creation fail");

    int nangles_device=(nangles+deviceCount-1)/deviceCount;
    int nangles_last_device=(nangles-(deviceCount-1)*nangles_device);
    unsigned int noOfKernelCalls = (nangles_device+PROJ_PER_BLOCK-1)/PROJ_PER_BLOCK;  // We'll take care of bounds checking inside the loop if nalpha is not divisible by PROJ_PER_BLOCK
    unsigned int noOfKernelCallsLastDev = (nangles_last_device+PROJ_PER_BLOCK-1)/PROJ_PER_BLOCK; // we will use this in the memory management.
    int projection_this_block;
    hipTextureObject_t *texImg = new hipTextureObject_t[deviceCount];
    hipArray **d_cuArrTex = new hipArray*[deviceCount];
    
    for (unsigned int sp=0;sp<splits;sp++){
        
        // Create texture objects for all GPUs
        
        
        size_t linear_idx_start;
        //First one should always be  the same size as all the rest but the last
        linear_idx_start= (size_t)sp*(size_t)geoArray[0].nVoxelX*(size_t)geoArray[0].nVoxelY*(size_t)geoArray[0].nVoxelZ;
        
        
        CreateTexture(gpuids,&img[linear_idx_start],geoArray[sp],d_cuArrTex,texImg,!sp);
        cudaCheckErrors("Texture object creation fail");
        
        
        // Prepare kernel lauch variables
        
        int divU,divV;
        divU=PIXEL_SIZE_BLOCK/8;
        divV=PIXEL_SIZE_BLOCK;
        dim3 blocks(ceil((2*geoArray[sp].gelTubeRadius/geoArray[sp].dVoxelY+divU-1)/divU),ceil((geoArray[sp].nDetecV+divV-1)/divV), 1);
        dim3 threadsPerBlock(divU,divV,PROJ_PER_BLOCK);
        
        unsigned int proj_global;
        // Now that we have prepared the image (piece of image) and parameters for kernels
        // we project for all angles.

        for (unsigned int i=0; i<noOfKernelCalls; i++) { // Iterating over all 720 projs / PROJ_PER_BLOCK
            for (dev=0;dev<deviceCount;dev++){
                hipSetDevice(gpuids[dev]);
                
                for(unsigned int j=0; j<PROJ_PER_BLOCK; j++){
                    proj_global=(i*PROJ_PER_BLOCK+j)+dev*nangles_device;
                    if (proj_global>=nangles)
                        break;
                    if ((i*PROJ_PER_BLOCK+j)>=nangles_device)
                        break;
                    geoArray[sp].alpha=angles[proj_global*3];
                    geoArray[sp].theta=angles[proj_global*3+1];
                    geoArray[sp].psi  =angles[proj_global*3+2];
                    Point3D aux, aux2;
                    aux.x = geoArray[sp].DSD[proj_global];
                    aux.y = geoArray[sp].DSO[proj_global];
                    aux.z = geoArray[sp].EPS;
                    aux2.x = geoArray[sp].gelTubeRadius;

                    //precomute distances for faster execution
                    //Precompute per angle constant stuff for speed
                    computeDeltas_Siddon(geoArray[sp], proj_global, &uvOrigin, &deltaU, &deltaV, &source);
                    //Ray tracing!
                    projParamsArrayHost[6 * j] = uvOrigin;		// 6*j because we have 6 Point3D values per projection
                    projParamsArrayHost[6 * j + 1] = deltaU;
                    projParamsArrayHost[6 * j + 2] = deltaV;
                    projParamsArrayHost[6 * j + 3] = source;
                    projParamsArrayHost[6 * j + 4] = aux;
                    projParamsArrayHost[6 * j + 5] = aux2;
                }
                hipMemcpyToSymbolAsync(HIP_SYMBOL(projParamsArrayDev), projParamsArrayHost, sizeof(Point3D)*6*PROJ_PER_BLOCK,0,hipMemcpyHostToDevice,stream[dev*2]);
                hipStreamSynchronize(stream[dev*2]);
                cudaCheckErrors("kernel fail");
                kernelPixelDetector<<<blocks,threadsPerBlock,0,stream[dev*2]>>>(geoArray[sp],dProjection[(i%2)+dev*2],i,nangles_device,texImg[dev]);
            }

            // Now that the computation is happening, we need to either prepare the memory for
            // combining of the projections (splits>1) and start removing previous results.
            
            
            // If our image does not fit in memory then we need to make sure we accumulate previous results too.
            // This is done in 2 steps: 
            // 1)copy previous results back into GPU 
            // 2)accumulate with current results
            // The code to take them out is the same as when there are no splits needed
            if( !fits_in_memory&&sp>0)
            {
                // 1) grab previous results and put them in the auxiliary variable dProjection_accum
                for (dev = 0; dev < deviceCount; dev++)
                {
                    hipSetDevice(gpuids[dev]);
                    //Global index of FIRST projection on this set on this GPU
                    proj_global=i*PROJ_PER_BLOCK+dev*nangles_device;
                    if(proj_global>=nangles) 
                        break;

                    // Unless its the last projection set, we have PROJ_PER_BLOCK angles. Otherwise...
                    if(i+1==noOfKernelCalls) //is it the last block?
                        projection_this_block=min(nangles_device-(noOfKernelCalls-1)*PROJ_PER_BLOCK, //the remaining angles that this GPU had to do (almost never PROJ_PER_BLOCK)
                                                  nangles-proj_global);                              //or whichever amount is left to finish all (this is for the last GPU)
                    else
                        projection_this_block=PROJ_PER_BLOCK;

                    hipMemcpyAsync(dProjection_accum[(i%2)+dev*2], result[proj_global], projection_this_block*geo.nDetecV*geo.nDetecU*sizeof(float), hipMemcpyHostToDevice,stream[dev*2+1]);
                }
                //  2) take the results from current compute call and add it to the code in execution.
                for (dev = 0; dev < deviceCount; dev++)
                {
                    hipSetDevice(gpuids[dev]);
                    //Global index of FIRST projection on this set on this GPU
                    proj_global=i*PROJ_PER_BLOCK+dev*nangles_device;
                    if(proj_global>=nangles) 
                        break;

                    // Unless its the last projection set, we have PROJ_PER_BLOCK angles. Otherwise...
                    if(i+1==noOfKernelCalls) //is it the last block?
                        projection_this_block=min(nangles_device-(noOfKernelCalls-1)*PROJ_PER_BLOCK, //the remaining angles that this GPU had to do (almost never PROJ_PER_BLOCK)
                                                  nangles-proj_global);                              //or whichever amount is left to finish all (this is for the last GPU)
                    else
                        projection_this_block=PROJ_PER_BLOCK;

                    hipStreamSynchronize(stream[dev*2+1]); // wait until copy is finished
                    vecAddInPlace<<<(geo.nDetecU*geo.nDetecV*projection_this_block+MAXTREADS-1)/MAXTREADS,MAXTREADS,0,stream[dev*2]>>>(dProjection[(i%2)+dev*2],dProjection_accum[(i%2)+dev*2],(unsigned long)geo.nDetecU*geo.nDetecV*projection_this_block);
                }
            } // end accumulation case, where the image needs to be split 

            // Now, lets get out the projections from the previous execution of the kernels.
            if (i>0){
                for (dev = 0; dev < deviceCount; dev++)
                {
                    hipSetDevice(gpuids[dev]);
                    //Global index of FIRST projection on previous set on this GPU
                    proj_global=(i-1)*PROJ_PER_BLOCK+dev*nangles_device;
                    if (dev+1==deviceCount) {    //is it the last device?
                        // projections assigned to this device is >=nangles_device-(deviceCount-1) and < nangles_device
                        if (i-1 < noOfKernelCallsLastDev) {
                            // The previous set(block) was not empty.
                            projection_this_block=min(PROJ_PER_BLOCK, nangles-proj_global);
                        }
                        else {
                            // The previous set was empty.
                            // This happens if deviceCount > PROJ_PER_BLOCK+1.
                            // e.g. PROJ_PER_BLOCK = 9, deviceCount = 11, nangles = 199.
                            // e.g. PROJ_PER_BLOCK = 1, deviceCount =  3, nangles =   7.
                            break;
                        }
                    }
                    else {
                        projection_this_block=PROJ_PER_BLOCK;
                    }
                    hipMemcpyAsync(result[proj_global], dProjection[(int)(!(i%2))+dev*2],  projection_this_block*geo.nDetecV*geo.nDetecU*sizeof(float), hipMemcpyDeviceToHost,stream[dev*2+1]);
                }
            }
            // Make sure Computation on kernels has finished before we launch the next batch.
            for (dev = 0; dev < deviceCount; dev++){
                hipSetDevice(gpuids[dev]);
                hipStreamSynchronize(stream[dev*2]);
            }
        }
        
         // We still have the last set of projections to get out of GPUs
        for (dev = 0; dev < deviceCount; dev++)
        {
            hipSetDevice(gpuids[dev]);
            //Global index of FIRST projection on this set on this GPU
            proj_global=(noOfKernelCalls-1)*PROJ_PER_BLOCK+dev*nangles_device;
            if(proj_global>=nangles) 
                break;
            // How many projections are left here?
            projection_this_block=min(nangles_device-(noOfKernelCalls-1)*PROJ_PER_BLOCK, //the remaining angles that this GPU had to do (almost never PROJ_PER_BLOCK)
                                      nangles-proj_global);                              //or whichever amount is left to finish all (this is for the last GPU)

            hipDeviceSynchronize(); //Not really necesary, but just in case, we los nothing. 
            cudaCheckErrors("Error at copying the last set of projections out (or in the previous copy)");
            hipMemcpyAsync(result[proj_global], dProjection[(int)(!(noOfKernelCalls%2))+dev*2], projection_this_block*geo.nDetecV*geo.nDetecU*sizeof(float), hipMemcpyDeviceToHost,stream[dev*2+1]);
        }
        // Make sure everyone has done their bussiness before the next image split:
        hipDeviceSynchronize();

    } // End image split loop.
    
    cudaCheckErrors("Main loop  fail");
    ///////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////

    for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(gpuids[dev]);
            hipDestroyTextureObject(texImg[dev]);
            hipFreeArray(d_cuArrTex[dev]);
    }
    delete[] texImg; texImg = 0;
    delete[] d_cuArrTex; d_cuArrTex = 0;
    // Freeing Stage
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(gpuids[dev]);
        hipFree(dProjection[dev*2]);
        hipFree(dProjection[dev*2+1]);
        
    }
    free(dProjection);
    
    if(!fits_in_memory){
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(gpuids[dev]);
            hipFree(dProjection_accum[dev*2]);
            hipFree(dProjection_accum[dev*2+1]);
            
        }
        free(dProjection_accum);
    }
    freeGeoArray(splits,geoArray);
    hipHostFree(projParamsArrayHost);
   
    
    for (int i = 0; i < nStreams; ++i)
        hipStreamDestroy(stream[i]) ;
#ifndef NO_PINNED_MEMORY
    if (isHostRegisterSupported & (splits>1 |deviceCount>1)){
        hipHostUnregister(img);
    }
    cudaCheckErrors("hipFree  fail");
#endif
    hipDeviceReset();
    return 0;
}




void CreateTexture(const GpuIds& gpuids,const float* imagedata,Geometry geo,hipArray** d_cuArrTex, hipTextureObject_t *texImage,bool alloc)
{
    //size_t size_image=geo.nVoxelX*geo.nVoxelY*geo.nVoxelZ;
    const hipExtent extent = make_hipExtent(geo.nVoxelX, geo.nVoxelY, geo.nVoxelZ);
    const unsigned int num_devices = gpuids.GetLength();
    if(alloc){
        for (unsigned int dev = 0; dev < num_devices; dev++){
            hipSetDevice(gpuids[dev]);
            
            //hipArray Descriptor
            hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
            //cuda Array
            hipMalloc3DArray(&d_cuArrTex[dev], &channelDesc, extent);
        }
    }
    for (unsigned int dev = 0; dev < num_devices; dev++){
        hipSetDevice(gpuids[dev]);
        hipMemcpy3DParms copyParams = {0};
        //Array creation
        copyParams.srcPtr   = make_hipPitchedPtr((void *)imagedata, extent.width*sizeof(float), extent.width, extent.height);
        copyParams.dstArray = d_cuArrTex[dev];
        copyParams.extent   = extent;
        copyParams.kind     = hipMemcpyHostToDevice;
        hipMemcpy3DAsync(&copyParams);
    }
    for (unsigned int dev = 0; dev < num_devices; dev++){
        hipSetDevice(gpuids[dev]);
        hipResourceDesc    texRes;
        memset(&texRes, 0, sizeof(hipResourceDesc));
        texRes.resType = hipResourceTypeArray;
        texRes.res.array.array  = d_cuArrTex[dev];
        hipTextureDesc     texDescr;
        memset(&texDescr, 0, sizeof(hipTextureDesc));
        texDescr.normalizedCoords = false;
        texDescr.filterMode = hipFilterModePoint;
        texDescr.addressMode[0] = hipAddressModeBorder;
        texDescr.addressMode[1] = hipAddressModeBorder;
        texDescr.addressMode[2] = hipAddressModeBorder;
        texDescr.readMode = hipReadModeElementType;
        hipCreateTextureObject(&texImage[dev], &texRes, &texDescr, NULL);
        
    }
    for (unsigned int dev = 0; dev < num_devices; dev++){
        hipSetDevice(gpuids[dev]);
        hipDeviceSynchronize();
    }
    cudaCheckErrors("Texture object creation fail");
}

/* This code generates the geometries needed to split the image properly in
 * cases where the entire image does not fit in the memory of the GPU
 **/
void splitImage(unsigned int splits,Geometry geo,Geometry* geoArray, unsigned int nangles){
    
    unsigned long splitsize=(geo.nVoxelZ+splits-1)/splits;// ceil if not divisible
    for(unsigned int sp=0;sp<splits;sp++){
        geoArray[sp]=geo;
        // All of them are splitsize, but the last one, possible
        geoArray[sp].nVoxelZ=((sp+1)*splitsize<geo.nVoxelZ)?  splitsize:  geo.nVoxelZ-splitsize*sp;
        geoArray[sp].sVoxelZ= geoArray[sp].nVoxelZ* geoArray[sp].dVoxelZ;
        
        // We need to redefine the offsets, as now each subimage is not aligned in the origin.
        geoArray[sp].offOrigZ=(float *)malloc(nangles*sizeof(float));
        for (unsigned int i=0;i<nangles;i++){
            geoArray[sp].offOrigZ[i]=geo.offOrigZ[i]-geo.sVoxelZ/2+sp*geoArray[0].sVoxelZ+geoArray[sp].sVoxelZ/2;
        }
        
    }
    
}

/* This code precomputes The location of the source and the Delta U and delta V (in the warped space)
 * to compute the locations of the x-rays. While it seems verbose and overly-optimized,
 * it does saves about 30% of each of the kernel calls. Thats something!
 **/
void computeDeltas_Siddon(Geometry geo,int i, Point3D* uvorigin, Point3D* deltaU, Point3D* deltaV, Point3D* source){

    
    Point3D S;
    S.x=geo.DSO[i];
    S.y=0;
    S.z=0;
    
    //End point
    Point3D P,Pu0,Pv0;
    
    P.x  =-(geo.DSD[i]-geo.DSO[i]);   P.y  = geo.dDetecU*(0-((float)geo.nDetecU/2)+0.5);       P.z  = geo.dDetecV*(((float)geo.nDetecV/2)-0.5-0);
    Pu0.x=-(geo.DSD[i]-geo.DSO[i]);   Pu0.y= geo.dDetecU*(1-((float)geo.nDetecU/2)+0.5);       Pu0.z= geo.dDetecV*(((float)geo.nDetecV/2)-0.5-0);
    Pv0.x=-(geo.DSD[i]-geo.DSO[i]);   Pv0.y= geo.dDetecU*(0-((float)geo.nDetecU/2)+0.5);       Pv0.z= geo.dDetecV*(((float)geo.nDetecV/2)-0.5-1);
    // Geomtric trasnformations:
    // Now we have the Real world (OXYZ) coordinates of the UPPER LEFT corner and its two neighbours.
    // The obkjective is to get a position of the detector in a coordinate system where:
    // 1-units are voxel size (in each direction can be different)
    // 2-The image has the its first voxel at (0,0,0)
    // 3-The image never rotates
    
    // To do that, we need to compute the "deltas" the detector, or "by how much
    // (in new xyz) does the voxels change when and index is added". To do that
    // several geometric steps needs to be changed
    
    //1.Roll,pitch,jaw
    // The detector can have a small rotation.
    // according to
    //"A geometric calibration method for cone beam CT systems" Yang K1, Kwan AL, Miller DF, Boone JM. Med Phys. 2006 Jun;33(6):1695-706.
    // Only the Z rotation will have a big influence in the image quality when they are small.
    // Still all rotations are supported
    
    // To roll pitch jaw, the detector has to be in centered in OXYZ.
    P.x=0;Pu0.x=0;Pv0.x=0;
    
    // Roll pitch yaw
    rollPitchYaw(geo,i,&P);
    rollPitchYaw(geo,i,&Pu0);
    rollPitchYaw(geo,i,&Pv0);
    //Now ltes translate the points where they should be:
    P.x=P.x-(geo.DSD[i]-geo.DSO[i]);
    Pu0.x=Pu0.x-(geo.DSD[i]-geo.DSO[i]);
    Pv0.x=Pv0.x-(geo.DSD[i]-geo.DSO[i]);
    
    //1: Offset detector
    
    
    //S doesnt need to chagne
    
    
    //3: Rotate (around z)!
    Point3D Pfinal, Pfinalu0, Pfinalv0;
    Pfinal.x  =P.x;
    Pfinal.y  =P.y  +geo.offDetecU[i]; Pfinal.z  =P.z  +geo.offDetecV[i];
    Pfinalu0.x=Pu0.x;
    Pfinalu0.y=Pu0.y  +geo.offDetecU[i]; Pfinalu0.z  =Pu0.z  +geo.offDetecV[i];
    Pfinalv0.x=Pv0.x;
    Pfinalv0.y=Pv0.y  +geo.offDetecU[i]; Pfinalv0.z  =Pv0.z  +geo.offDetecV[i];
    
    eulerZYZ(geo,&Pfinal);
    eulerZYZ(geo,&Pfinalu0);
    eulerZYZ(geo,&Pfinalv0);
    eulerZYZ(geo,&S);
    
    //2: Offset image (instead of offseting image, -offset everything else)
    
    Pfinal.x  =Pfinal.x-geo.offOrigX[i];     Pfinal.y  =Pfinal.y-geo.offOrigY[i];     Pfinal.z  =Pfinal.z-geo.offOrigZ[i];
    Pfinalu0.x=Pfinalu0.x-geo.offOrigX[i];   Pfinalu0.y=Pfinalu0.y-geo.offOrigY[i];   Pfinalu0.z=Pfinalu0.z-geo.offOrigZ[i];
    Pfinalv0.x=Pfinalv0.x-geo.offOrigX[i];   Pfinalv0.y=Pfinalv0.y-geo.offOrigY[i];   Pfinalv0.z=Pfinalv0.z-geo.offOrigZ[i];
    S.x=S.x-geo.offOrigX[i];               S.y=S.y-geo.offOrigY[i];               S.z=S.z-geo.offOrigZ[i];
    
    // As we want the (0,0,0) to be in a corner of the image, we need to translate everything (after rotation);
    Pfinal.x  =Pfinal.x+geo.sVoxelX/2;      Pfinal.y  =Pfinal.y+geo.sVoxelY/2;          Pfinal.z  =Pfinal.z  +geo.sVoxelZ/2;
    Pfinalu0.x=Pfinalu0.x+geo.sVoxelX/2;    Pfinalu0.y=Pfinalu0.y+geo.sVoxelY/2;        Pfinalu0.z=Pfinalu0.z+geo.sVoxelZ/2;
    Pfinalv0.x=Pfinalv0.x+geo.sVoxelX/2;    Pfinalv0.y=Pfinalv0.y+geo.sVoxelY/2;        Pfinalv0.z=Pfinalv0.z+geo.sVoxelZ/2;
    S.x      =S.x+geo.sVoxelX/2;          S.y      =S.y+geo.sVoxelY/2;              S.z      =S.z      +geo.sVoxelZ/2;
    
    //4. Scale everything so dVoxel==1
    Pfinal.x  =Pfinal.x/geo.dVoxelX;      Pfinal.y  =Pfinal.y/geo.dVoxelY;        Pfinal.z  =Pfinal.z/geo.dVoxelZ;
    Pfinalu0.x=Pfinalu0.x/geo.dVoxelX;    Pfinalu0.y=Pfinalu0.y/geo.dVoxelY;      Pfinalu0.z=Pfinalu0.z/geo.dVoxelZ;
    Pfinalv0.x=Pfinalv0.x/geo.dVoxelX;    Pfinalv0.y=Pfinalv0.y/geo.dVoxelY;      Pfinalv0.z=Pfinalv0.z/geo.dVoxelZ;
    S.x      =S.x/geo.dVoxelX;          S.y      =S.y/geo.dVoxelY;            S.z      =S.z/geo.dVoxelZ;
    
    
    //mexPrintf("COR: %f \n",geo.COR[i]);
    //5. apply COR. Wherever everything was, now its offesetd by a bit
    float CORx, CORy;
    CORx=-geo.COR[i]*sin(geo.alpha)/geo.dVoxelX;
    CORy= geo.COR[i]*cos(geo.alpha)/geo.dVoxelY;
    Pfinal.x+=CORx;   Pfinal.y+=CORy;
    Pfinalu0.x+=CORx;   Pfinalu0.y+=CORy;
    Pfinalv0.x+=CORx;   Pfinalv0.y+=CORy;
    S.x+=CORx; S.y+=CORy;
    
    // return
    
    *uvorigin=Pfinal;
    
    deltaU->x=Pfinalu0.x-Pfinal.x;
    deltaU->y=Pfinalu0.y-Pfinal.y;
    deltaU->z=Pfinalu0.z-Pfinal.z;
    
    deltaV->x=Pfinalv0.x-Pfinal.x;
    deltaV->y=Pfinalv0.y-Pfinal.y;
    deltaV->z=Pfinalv0.z-Pfinal.z;
    
    *source=S;
}


#ifndef PROJECTION_HPP

float maxDistanceCubeXY(Geometry geo, float alpha,int i){
    ///////////
    // Compute initial "t" so we access safely as less as out of bounds as possible.
    //////////
    
    
    float maxCubX,maxCubY;
    // Forgetting Z, compute max distance: diagonal+offset
    maxCubX=(geo.sVoxelX/2+ abs(geo.offOrigX[i]))/geo.dVoxelX;
    maxCubY=(geo.sVoxelY/2+ abs(geo.offOrigY[i]))/geo.dVoxelY;
    
    return geo.DSO[i]/geo.dVoxelX-sqrt(maxCubX*maxCubX+maxCubY*maxCubY);
    
}
void rollPitchYaw(Geometry geo,int i, Point3D* point){
    Point3D auxPoint;
    auxPoint.x=point->x;
    auxPoint.y=point->y;
    auxPoint.z=point->z;
    
    point->x=cos(geo.dRoll[i])*cos(geo.dPitch[i])*auxPoint.x
            +(cos(geo.dRoll[i])*sin(geo.dPitch[i])*sin(geo.dYaw[i]) - sin(geo.dRoll[i])*cos(geo.dYaw[i]))*auxPoint.y
            +(cos(geo.dRoll[i])*sin(geo.dPitch[i])*cos(geo.dYaw[i]) + sin(geo.dRoll[i])*sin(geo.dYaw[i]))*auxPoint.z;
    
    point->y=sin(geo.dRoll[i])*cos(geo.dPitch[i])*auxPoint.x
            +(sin(geo.dRoll[i])*sin(geo.dPitch[i])*sin(geo.dYaw[i]) + cos(geo.dRoll[i])*cos(geo.dYaw[i]))*auxPoint.y
            +(sin(geo.dRoll[i])*sin(geo.dPitch[i])*cos(geo.dYaw[i]) - cos(geo.dRoll[i])*sin(geo.dYaw[i]))*auxPoint.z;
    
    point->z=-sin(geo.dPitch[i])*auxPoint.x
            +cos(geo.dPitch[i])*sin(geo.dYaw[i])*auxPoint.y
            +cos(geo.dPitch[i])*cos(geo.dYaw[i])*auxPoint.z;
    
}
void eulerZYZ(Geometry geo, Point3D* point){
    Point3D auxPoint;
    auxPoint.x=point->x;
    auxPoint.y=point->y;
    auxPoint.z=point->z;
    
    point->x=(+cos(geo.alpha)*cos(geo.theta)*cos(geo.psi)-sin(geo.alpha)*sin(geo.psi))*auxPoint.x+
            (-cos(geo.alpha)*cos(geo.theta)*sin(geo.psi)-sin(geo.alpha)*cos(geo.psi))*auxPoint.y+
            cos(geo.alpha)*sin(geo.theta)*auxPoint.z;
    
    point->y=(+sin(geo.alpha)*cos(geo.theta)*cos(geo.psi)+cos(geo.alpha)*sin(geo.psi))*auxPoint.x+
            (-sin(geo.alpha)*cos(geo.theta)*sin(geo.psi)+cos(geo.alpha)*cos(geo.psi))*auxPoint.y+
            sin(geo.alpha)*sin(geo.theta)*auxPoint.z;
    
    point->z=-sin(geo.theta)*cos(geo.psi)*auxPoint.x+
            sin(geo.theta)*sin(geo.psi)*auxPoint.y+
            cos(geo.theta)*auxPoint.z;
    
    
}
//______________________________________________________________________________
//
//      Function:       freeGeoArray
//
//      Description:    Frees the memory from the geometry array for multiGPU.
//______________________________________________________________________________
void freeGeoArray(unsigned int splits,Geometry* geoArray){
    for(unsigned int sp=0;sp<splits;sp++){
        free(geoArray[sp].offOrigZ);
    }
    free(geoArray);
}
//______________________________________________________________________________
//
//      Function:       checkFreeMemory
//
//      Description:    check available memory on devices
//______________________________________________________________________________
void checkFreeMemory(const GpuIds& gpuids, size_t *mem_GPU_global){
    size_t memfree;
    size_t memtotal;
    const int deviceCount = gpuids.GetLength();

    for (int dev = 0; dev < deviceCount; dev++){
        hipSetDevice(gpuids[dev]);
        hipMemGetInfo(&memfree,&memtotal);
        if(dev==0) *mem_GPU_global=memfree;
        if(memfree<memtotal/2){
            mexErrMsgIdAndTxt("Ax:Siddon_projection:GPUmemory","One (or more) of your GPUs is being heavily used by another program (possibly graphics-based).\n Free the GPU to run TIGRE\n");
        }
        cudaCheckErrors("Check mem error");
        
        *mem_GPU_global=(memfree<*mem_GPU_global)?memfree:*mem_GPU_global;
    }
    *mem_GPU_global=(size_t)((double)*mem_GPU_global*0.95);
    
    //*mem_GPU_global= insert your known number here, in bytes.
}
#endif
